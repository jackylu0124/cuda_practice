#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

#include <hip/hip_runtime.h>



float* readImage(int* width, int* height, std::string path) {
	std::ifstream reader;
	reader.open(path);
	if (!reader.is_open()) {
		std::cerr << "Unable to open file." << std::endl;
		exit(1);
	}

	// Read file content
	// Read magic number
	std::string magicNumber;
	reader >> magicNumber;
	if (magicNumber != "P3") {
		std::cerr << "The magic number is not P3 (The file is not an ASCII PPM file)." << std::endl;
		exit(1);
	}

	// Read width and height
	std::string widthStr, heightStr;
	reader >> widthStr >> heightStr;
	*width = stoi(widthStr);
	*height = stoi(heightStr);
	std::cout << "The image dimension is " << *width << " x " << *height << "." << std::endl;

	// Read range
	std::string rangeStr;
	float range;
	reader >> rangeStr;
	range = stof(rangeStr);

	// Read pixel values of the image, allocate a vector, and store the image's pixel values in the vector
	int n = *width * *height * 3;
	float* vec = (float*) malloc(n * sizeof(float));
	if (vec == NULL) {
		std::cout << "Malloc for vector in readImage(int* width, int* height, std::string path) fails." << std::endl;
		exit(1);
	}
	std::string pixelValStr;
	for (int i = 0; i < n; i++) {
		reader >> pixelValStr;
		vec[i] = stof(pixelValStr);
	}

	// Close the file
	reader.close();

	return vec;
}

void writeImage(float* vec, int width, int height, std::string path) {
	std::ofstream writer;
	writer.open(path);
	if (!writer.is_open()) {
		std::cerr << "Unable to write file." << std::endl;
		exit(1);
	}

	writer << "P3\n";
	writer << width << " " << height << "\n";
	writer << "255\n";
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) {
			for (int c = 0; c < 3; c++) {
				int index = (row * width + col) * 3 + c;
				writer << (int) vec[index] << " ";
			}
		}
		writer << "\n";
	}

	writer.close();
}

void colorToGrayCPU(std::string readPath, std::string writePath) {
	int width, height;
	float* inVec = readImage(&width, &height, readPath);
	float* outVec = (float*) malloc(width * height * 3 * sizeof(float));
	if (outVec == NULL) {
		std::cout << "Malloc for vector in colorToGrayCPU(std::string readPath, std::string writePath) fails." << std::endl;
		exit(1);
	}
	
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) {
			int index = (row * width + col) * 3;
			float r = inVec[index];
			float g = inVec[index + 1];
			float b = inVec[index + 2];

			float gray = 0.21 * r + 0.72 * g + 0.07 * b;
			outVec[index] = gray;
			outVec[index + 1] = gray;
			outVec[index + 2] = gray;
		}
	}
	writeImage(outVec, width, height, writePath);

	// Free allocated memories
	free(outVec);
	free(inVec);
}

int main() {
	// TODO
	std::string readPath = "pineapple_pizza.ppm";
	std::string writePath = "pineapple_pizza_gray_cpu.ppm";
	colorToGrayCPU(readPath, writePath);
	return 0;
}